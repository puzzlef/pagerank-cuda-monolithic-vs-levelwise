#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>
#include <iostream>
#include <algorithm>
#include "src/main.hxx"

using namespace std;




#define REPEAT 5

template <class G, class H>
void runPagerank(const G& x, const H& xt, bool show) {
  vector<double> *init = nullptr;

  // Find pagerank using nvGraph.
  auto a1 = pagerankNvgraph(xt, init, {REPEAT});
  auto e1 = l1Norm(a1.ranks, a1.ranks);
  printf("[%09.3f ms; %03d iters.] [%.4e err.] pagerankNvgraph\n", a1.time, a1.iterations, e1);
  if (show) println(a1.ranks);

  // Find pagerank using CUDA (monolithic).
  auto a2 = pagerankMonolithic(xt, init, {REPEAT});
  auto e2 = l1Norm(a2.ranks, a1.ranks);
  printf("[%09.3f ms; %03d iters.] [%.4e err.] pagerankMonolithic\n", a2.time, a2.iterations, e2);

  // Find levelwise pagerank using CUDA, adjusting min. component size.
  for (int C=100, i=0; C<=max(x.order(), 100); C*=i&1? 2:5, i++) {
    auto a3 = pagerankLevelwise(x, xt, init, {REPEAT, C});
    auto e3 = l1Norm(a3.ranks, a1.ranks);
    printf("[%09.3f ms; %03d iters.] [%.4e err.] pagerankLevelwise [%.0e min-component-size]\n", a3.time, a3.iterations, e3, (double) C);
  }
}


int main(int argc, char **argv) {
  char *file = argv[1];
  bool  show = argc > 2;
  printf("Loading graph %s ...\n", file);
  auto x  = readMtx(file); println(x);
  loopDeadEnds(x); print(x); printf(" (loopDeadEnds)\n");
  auto xt = transposeWithDegree(x); print(xt); printf(" (transposeWithDegree)\n");
  runPagerank(x, xt, show);
  printf("\n");
  return 0;
}
