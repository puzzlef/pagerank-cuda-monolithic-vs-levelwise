#include "hip/hip_runtime.h"
#include <cmath>
#include <vector>
#include <string>
#include <cstdio>
#include <iostream>
#include "src/main.hxx"

using namespace std;




template <class G, class H>
void runPagerank(const G& x, const H& xt, int repeat) {
  enum NormFunction { L0=0, L1=1, L2=2, Li=3 };
  vector<float> *init = nullptr;

  // Componentwise PageRank data.
  auto cs = sortedComponents(x, xt);
  PagerankData<G> D {move(cs), G()};

  // Find pagerank using L1-norm for convergence check.
  auto a1 = pagerankMonolithicCuda(x, xt, init, {repeat, L1});
  auto e1 = l1Norm(a1.ranks, a1.ranks);
  printf("[%09.3f ms; %03d iters.] [%.4e err.] pagerankMonolithicCudaL1Norm\n", a1.time, a1.iterations, e1);
  for (int MC=1, i=0; MC<=5e+7; MC*=i&1? 2:5, i++) {
    auto a2 = pagerankComponentwiseCuda(x, xt, init, {repeat, L1, MC}, D);
    auto e2 = l1Norm(a2.ranks, a1.ranks);
    printf("[%09.3f ms; %03d iters.] [%.4e err.] pagerankComponentwiseCudaL1Norm [min-compute=%d]\n", a2.time, a2.iterations, e2, MC);
  }

  // Find pagerank using L2-norm for convergence check.
  auto a3 = pagerankMonolithicCuda(x, xt, init, {repeat, L2});
  auto e3 = l1Norm(a3.ranks, a1.ranks);
  printf("[%09.3f ms; %03d iters.] [%.4e err.] pagerankMonolithicCudaL2Norm\n", a3.time, a3.iterations, e3);
  for (int MC=1, i=0; MC<=5e+7; MC*=i&1? 2:5, i++) {
    auto a4 = pagerankComponentwiseCuda(x, xt, init, {repeat, L2, MC}, D);
    auto e4 = l1Norm(a4.ranks, a1.ranks);
    printf("[%09.3f ms; %03d iters.] [%.4e err.] pagerankComponentwiseCudaL2Norm [min-compute=%d]\n", a4.time, a4.iterations, e4, MC);
  }

  // Find pagerank using Li-norm for convergence check.
  auto a5 = pagerankMonolithicCuda(x, xt, init, {repeat, Li});
  auto e5 = l1Norm(a5.ranks, a1.ranks);
  printf("[%09.3f ms; %03d iters.] [%.4e err.] pagerankMonolithicCudaLiNorm\n", a5.time, a5.iterations, e5);
  for (int MC=1, i=0; MC<=5e+7; MC*=i&1? 2:5, i++) {
    auto a6 = pagerankComponentwiseCuda(x, xt, init, {repeat, Li, MC}, D);
    auto e6 = l1Norm(a6.ranks, a1.ranks);
    printf("[%09.3f ms; %03d iters.] [%.4e err.] pagerankComponentwiseCudaLiNorm [min-compute=%d]\n", a6.time, a6.iterations, e6, MC);
  }
}


int main(int argc, char **argv) {
  char *file = argv[1];
  int repeat = argc>2? stoi(argv[2]) : 5;
  printf("Loading graph %s ...\n", file);
  auto x  = readMtx(file); println(x);
  // Handle dead ends with loop strategy (alternatives: loop-all, remove).
  selfLoopTo(x, [&](int u) { return isDeadEnd(x, u); });
  print(x); printf(" (selfLoopDeadEnds)\n");
  // Transpose graph after handling dead ends.
  auto xt = transposeWithDegree(x); print(xt); printf(" (transposeWithDegree)\n");
  runPagerank(x, xt, repeat);
  printf("\n");
  return 0;
}
